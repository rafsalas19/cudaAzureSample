#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <iostream>;
#include <fstream>
#include <string>

using namespace std;

#define ITERATION 500
#define LENX 100
#define LENY 100

void checkError(hipError_t err) {
	if (err != hipSuccess)
	{
		fprintf(stderr, "Cuda error (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

}

__global__ void
transferHeat(const float* inGrid, float* outGrid)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i > LENX * LENY) {
		return;
	}



	if (i > LENX && i <(LENX * LENY -1)-LENX)
	{		
		outGrid[i] = .25 * (inGrid[i-1] + inGrid[i+1] + inGrid[i + LENY] + inGrid[i - LENY]);
		//if(i==29)printf("%f | %f %f %f %f\n",outGrid[i], inGrid[i - 1], inGrid[i + 1], inGrid[i + LENY], inGrid[i - LENY]);
	}
	else
	{
		outGrid[i] = inGrid[i];
	}
}

void default_initialize(float * gridPointsCPU, int lenX, int lenY) {
	for (int j = 0;j<lenY;++j) {
		for (int i = 0; i < lenX; ++i) {
			int point = i + j * lenX;
			if (j == 0) {
				gridPointsCPU[point] = 100;
			}

			else if (j == lenY-1) {
				gridPointsCPU[point] = 0;
			}
			else if (j ==0 && i%lenX==0) {
				gridPointsCPU[point] = 0;
			}
			else if (j == lenY - 1 && i % lenX == lenX-1) {
				gridPointsCPU[point] = 0;
			}
			else {
				gridPointsCPU[point] = 30;
			}
		}
	}

	return;
}


int main(){
	int probSize = LENX * LENY;	
	float gridPointsCPU[LENX * LENY] = {0};
	float* gridPointsOutGPU = NULL;
	float* gridPointsInGPU = NULL;
	int devicecount = 0;

	//check device count
	hipGetDeviceCount(&devicecount);
	cout << "Device count is " << devicecount << endl;
	hipError_t err = hipSuccess;

	err = hipMalloc((void**)&gridPointsInGPU, probSize * sizeof(float));
	checkError(err);
	err = hipMalloc((void**)&gridPointsOutGPU, probSize * sizeof(float));
	checkError(err);

	//initialize
	default_initialize(gridPointsCPU,LENX,LENY);

	//copy from cpu
	err = hipMemcpy(gridPointsInGPU, gridPointsCPU, probSize*sizeof(float), hipMemcpyHostToDevice);
	checkError(err);

	//fstream fstartBuffer;
	//fstartBuffer.open("htStart", fstream::out | fstream::trunc);
	//for (int i = 0; i < probSize; ++i) {
	//	//fstartBuffer << gridPointsCPU[i] << endl;
	//	int x = i % LENX;
	//	int y = i / LENY;
	//	fstartBuffer << "index: " << i << " point:" << x << "," << y << " " << gridPointsCPU[i] << endl;
	//}
	//fstartBuffer.close();


	int gridSize = 0;
	int blockSize = 128;
	gridSize = probSize/blockSize +1;
	for (int i = 0; i < ITERATION ; ++i) {



		transferHeat << < gridSize, blockSize >> > (gridPointsInGPU, gridPointsOutGPU);
		hipDeviceSynchronize();

		if (i <( ITERATION - 1)) {
			float* swap = gridPointsInGPU;
			gridPointsInGPU = gridPointsOutGPU;
			gridPointsOutGPU = swap;
		}
	}
	
	err = hipGetLastError();
	checkError(err);
	err = hipMemcpy(gridPointsCPU, gridPointsOutGPU, probSize * sizeof(float), hipMemcpyDeviceToHost);
	checkError(err);

	fstream fout;
	fout.open("htOut", fstream::out | fstream::trunc);
	for (int i = 0;i<probSize;++i) {
		int x = i % LENX;
		int y = i / LENY;
		fout  << x <<","<<y<<" "<<gridPointsCPU[i] << endl;
	}
	fout.close();
//	char endChar = 'c';
//	cout << "enter a character to exit"<<endl;
//	cin >> endChar;

	return 0;
}
